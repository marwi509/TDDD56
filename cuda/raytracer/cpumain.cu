#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */


#include "hip/hip_runtime.h"
#include "book.h"
#include "cpu_bitmap.h"
#define DIM 1024

#define rnd( x ) (x * rand() / RAND_MAX)
#define INF     2e10f

struct Sphere {
    float   r,b,g;
    float   radius;
    float   x,y,z;
    float hit( float ox, float oy, float *n ) {
        float dx = ox - x;
        float dy = oy - y;
        if (dx*dx + dy*dy < radius*radius) {
            float dz = sqrtf( radius*radius - dx*dx - dy*dy );
            *n = dz / sqrtf( radius * radius );
            return dz + z;
        }
        return -INF;
    }
};
#define SPHERES 500


void kernel(int i,int j, Sphere *s, unsigned char *ptr ) {
    // map from threadIdx/BlockIdx to pixel position
    int x = i; //threadIdx.x + blockIdx.x * blockDim.x;
    int y = j; //threadIdx.y + blockIdx.y * blockDim.y;
    //int offset = x + y * blockDim.x * gridDim.x;
    int offset = x + y*DIM;// * DIM * (DIM/16);
    
    float   ox = (x - DIM/2);
    float   oy = (y - DIM/2);

    float   r=0, g=0, b=0;
    float   maxz = -INF;
    for(int i=0; i<SPHERES; i++) {
        float   n;
        float   t = s[i].hit( ox, oy, &n );
        if (t > maxz) {
            float fscale = n;
            r = s[i].r * fscale;
            g = s[i].g * fscale;
            b = s[i].b * fscale;
            maxz = t;
        }
    } 

    ptr[offset*4 + 0] = (int)(r * 255);
    ptr[offset*4 + 1] = (int)(g * 255);
    ptr[offset*4 + 2] = (int)(b * 255);
    ptr[offset*4 + 3] = 255;
}



int main( void ) {
    
    CPUBitmap bitmap( DIM, DIM );


    // allocate temp memory, initialize it, copy to
    // memory on the GPU, then free our temp memory
    Sphere *temp_s = (Sphere*)malloc( sizeof(Sphere) * SPHERES );
    for (int i=0; i<SPHERES; i++) {
        temp_s[i].r = rnd( 1.0f );
        temp_s[i].g = rnd( 1.0f );
        temp_s[i].b = rnd( 1.0f );
        temp_s[i].x = rnd( 1000.0f ) - 500;
        temp_s[i].y = rnd( 1000.0f ) - 500;
        temp_s[i].z = rnd( 1000.0f ) - 500;
        temp_s[i].radius = rnd( 100.0f ) + 20;
    }
   
    // generate a bitmap from our sphere data
    dim3    grids(DIM/16,DIM/16);
    dim3    threads(16,16);
    
    unsigned char *pbitmap = bitmap.get_ptr();
   // pbitmap = malloc(  bitmap.image_size() );
    
    for(int i=0;i<DIM;i++)
		for(int j=0;j<DIM;j++)
			kernel(i,j, temp_s, pbitmap );

   memcpy( bitmap.get_ptr(), pbitmap,
                              bitmap.image_size() );
   
free(temp_s);

    // display
    //bitmap.display_and_exit();
}

